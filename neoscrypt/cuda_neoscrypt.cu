#include "hip/hip_runtime.h"
// Neoscrypt Kernel by djm34 enchanced by Sp_ and Pallas (@bitcointalk and github)

#include <stdio.h>
#include <memory.h>
#include "cuda_vector.h" 
 
//extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

__device__ uint4 *W;
uint32_t *d_NNonce[MAX_GPUS];
__constant__ uint32_t pTarget[1];
__constant__ uint32_t input_init[16];
__constant__ uint32_t c_data[20];

#define BLAKE2S_BLOCK_SIZE    64U 
#define BLAKE2S_OUT_SIZE      32U
#define BLAKE2S_KEY_SIZE      32U
#define FASTKDF_BUFFER_SIZE  256U


static const __constant__ uint8 BLAKE2S_IV_Vec =
{
	0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
	0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
};

static const uint8 BLAKE2S_IV_Vechost =
{
	0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
	0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
};

static const __constant__ uint32_t BLAKE2S_SIGMA[10][16] =
{
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
};

static const uint32_t BLAKE2S_SIGMA_host[10][16] =
{
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
};


#define BLAKE_G(idx0, idx1, a, b, c, d, key) { \
	a += key[BLAKE2S_SIGMA[idx0][idx1]] + b; \
	d = __byte_perm(d^a, 0, 0x1032); \
	c += d; \
	b = rotateR(b^c, 12); \
	a += key[BLAKE2S_SIGMA[idx0][idx1+1]] + b; \
	d = __byte_perm(d^a, 0, 0x0321); \
	c += d; \
	b = rotateR(b^c, 7); \
} 

#define BLAKE_G_PRE(idx0,idx1, a, b, c, d, key) { \
	a += key[idx0] + b; \
	d = __byte_perm(d^a, 0, 0x1032); \
	c += d; \
	b = rotateR(b^c, 12); \
	a += key[idx1] + b; \
	d = __byte_perm(d^a, 0, 0x0321); \
	c += d; \
	b = rotateR(b^c, 7); \
} 

#define BLAKE_Ghost(idx0, idx1, a, b, c, d, key) { \
	idx = BLAKE2S_SIGMA_host[idx0][idx1]; a += key[idx]; \
	a += b; d = ROTR32(d^a,16); \
	c += d; b = ROTR32(b^c, 12); \
	idx = BLAKE2S_SIGMA_host[idx0][idx1+1]; a += key[idx]; \
	a += b; d = ROTR32(d^a,8); \
	c += d; b = ROTR32(b^c, 7); \
} 


static __forceinline__ __device__ void Blake2S(uint32_t* inout, const uint32_t* TheKey)
{
	uint16 V;
	uint8 tmpblock;
 					
	V.hi = BLAKE2S_IV_Vec; 
  V.lo = V.hi;
	V.lo.s0 ^= 0x01012020;

	// Copy input block for later
	tmpblock = V.lo;

	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;

//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(10,11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

//	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	BLAKE_G_PRE(9, 0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(5, 7, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(10, 15, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(14, 1, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(11, 12, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(6, 8, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(3, 13, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

//	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	BLAKE_G_PRE(2, 12, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(6, 10, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(0, 11, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(8, 3, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(4, 13, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(7, 5, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(15, 14, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(1, 9, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

//	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	BLAKE_G_PRE(12, 5, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(1, 15, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(14, 13, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(4, 10, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 7, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(6, 3, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(9, 2, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(8, 11, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

//	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	BLAKE_G_PRE(13, 11, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(7, 14, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(12, 1, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(3, 9, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(5, 0, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(15, 4, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(8, 6, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 10, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

//	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	BLAKE_G_PRE(6, 15, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(14, 9, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(11, 3, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(0, 8, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(12, 2, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(13, 7, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(1, 4, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(10, 5, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

//	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	BLAKE_G_PRE(10, 2, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(8, 4, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(7, 6, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(1, 5, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(15, 11, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(9, 14, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(3, 12, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(13, 0, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	V.lo ^= V.hi ^ tmpblock;
	V.hi = BLAKE2S_IV_Vec;
	tmpblock = V.lo;

	V.hi.s4 ^= 128;
	V.hi.s6 ^= 0xFFFFFFFF;

	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

	//		{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

	//		{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

	//		{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

//	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	BLAKE_G_PRE(9, 0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(5, 7, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(2, 4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(10, 15, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(14, 1, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(11, 12, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(6, 8, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(3, 13, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
/*
//	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	BLAKE_G_PRE(2, 12, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(6, 10, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(0, 11, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(8, 3, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(4, 13, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(7, 5, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(15, 14, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(1, 9, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

//	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	BLAKE_G_PRE(12, 5, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(1, 15, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(14, 13, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(4, 10, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(0, 7, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(6, 3, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(9, 2, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(8, 11, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

//	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	BLAKE_G_PRE(13, 11, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(7, 14, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(12, 1, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(3, 9, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(5, 0, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(15, 4, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(8, 6, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(2, 10, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

//	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	BLAKE_G_PRE(6, 15, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(14, 9, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(11, 3, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(0, 8, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(12, 2, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(13, 7, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(1, 4, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(10, 5, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

//	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	BLAKE_G_PRE(10, 2, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(8, 4, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(7, 6, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(1, 5, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(15, 11, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(9, 14, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(3, 12, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(13, 0, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
*/
	#pragma nounroll
	for (int x = 5; x < 10; ++x)
	{
		BLAKE_G(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
		BLAKE_G(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
		BLAKE_G(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
		BLAKE_G(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
		BLAKE_G(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
		BLAKE_G(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
		BLAKE_G(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
		BLAKE_G(x, 0x0E, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	}

	((uint8*)inout)[0] = V.lo ^ V.hi ^ tmpblock;
}


static __forceinline__ __host__ void Blake2Shost(uint32_t * inout, const uint32_t * inkey)
{
	uint16 V;
	uint32_t idx;
	uint8 tmpblock;

	V.hi = BLAKE2S_IV_Vechost;
	V.lo = BLAKE2S_IV_Vechost;
	V.lo.s0 ^= 0x01012020;

	tmpblock = V.lo;

	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;

	for (int x = 0; x < 10; ++x)
	{
		BLAKE_Ghost(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inkey);
		BLAKE_Ghost(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inkey);
		BLAKE_Ghost(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inkey);
		BLAKE_Ghost(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inkey);
		BLAKE_Ghost(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inkey);
		BLAKE_Ghost(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inkey);
		BLAKE_Ghost(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inkey);
		BLAKE_Ghost(x, 0x0E, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inkey);
	}

	V.lo ^= V.hi;
	V.lo ^= tmpblock;

	V.hi = BLAKE2S_IV_Vechost;
	tmpblock = V.lo;

	V.hi.s4 ^= 128;
	V.hi.s6 = ~V.hi.s6;

	for (int x = 0; x < 10; ++x)
	{
		BLAKE_Ghost(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
		BLAKE_Ghost(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
		BLAKE_Ghost(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
		BLAKE_Ghost(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
		BLAKE_Ghost(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
		BLAKE_Ghost(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
		BLAKE_Ghost(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
		BLAKE_Ghost(x, 0x0E, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	}

	V.lo ^= V.hi ^ tmpblock;

	((uint8*)inout)[0] = V.lo;
}


static __forceinline__ __device__ void shift256R2_final(uint32_t *ret, const uint8 &vec4, const uint32_t shift)
{
	uint32_t truc = 0, truc2 = cuda_swab32(vec4.s7), truc3 = 0;
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(truc) : "r"(truc3), "r"(truc2), "r"(shift));
	ret[8] = cuda_swab32(truc);
	truc3 = cuda_swab32(vec4.s6);
	asm("shf.r.clamp.b32 %0, %1, %2, %3;" : "=r"(truc) : "r"(truc2), "r"(truc3), "r"(shift));
	ret[7] = cuda_swab32(truc);
}


static __forceinline__ __device__ uint32_t fastkdf(const uint32_t *password, uint8_t *output, const uint32_t *salt)
{ 
	uint8_t bufidx, A[320], B[288];
	((uintx64*)A)[0] = ((uintx64*)password)[0];	// 256 bits
	((uint816*)A)[4] = ((uint816*)A)[0];	// 64 bits
	uint32_t input[BLAKE2S_BLOCK_SIZE/4];
	uint32_t key[BLAKE2S_BLOCK_SIZE / 4]={0};
	
	if (salt != NULL) {
		((uintx64*)B)[0] = ((uintx64*)salt)[0];
	 	((uint48 *)B)[8] = ((uint48 *)B)[0];
		((uint816*)input)[0] = ((uint816*)A)[0];
		((uint48*)key)[0] = ((uint48*)B)[0];
	} else {
		((uintx64*)B)[0] = ((uintx64*)A)[0];
		((uint48 *)B)[8] = ((uint48 *)A)[0];	// 32 bits
		((uint816*)input)[0] = ((uint816*)input_init)[0];
		((uint48*)key)[0] = ((uint48*)input)[0];
	}
	
	#pragma nounroll
	for (int i = 0; i < 32; ++i)
	{
		if (salt != NULL) Blake2S((uint32_t*)input, key);
		
		//const uchar4 bufhelper = ((uchar4*)input)[0] + ((uchar4*)input)[1] + ((uchar4*)input)[2] + ((uchar4*)input)[3] + ((uchar4*)input)[4] + ((uchar4*)input)[5] + ((uchar4*)input)[6] + ((uchar4*)input)[7];
		uchar4 bufhelper = ((uchar4*)input)[0];
		#pragma unroll
		for (int x = 1; x < BLAKE2S_OUT_SIZE / 4; ++x) bufhelper += ((uchar4*)input)[x];
		bufidx = bufhelper.x + bufhelper.y + bufhelper.z + bufhelper.w;
		uint32_t shifted[9];
  
		if (i == 31 && salt != NULL) {
			shift256R2_final(shifted, ((uint8*)input)[0], (bufidx & 3) << 3);
			((uint32_t *)B)[7 + (bufidx >> 2)] ^= shifted[7];
			((uint32_t *)B)[8 + (bufidx >> 2)] ^= shifted[8];
			break;
		}

		shift256R2(shifted, ((uint8*)input)[0], (bufidx & 3) << 3);
		#pragma unroll
		for (int k = 0; k < 9; ++k) ((uint32_t *)B)[k + (bufidx >> 2)] ^= shifted[k];

		if (bufidx < BLAKE2S_KEY_SIZE)                          ((uint8*)B)[8] = ((uint8*)B)[0];
		else if (bufidx > FASTKDF_BUFFER_SIZE-BLAKE2S_OUT_SIZE) ((uint8*)B)[0] = ((uint8*)B)[8];

		if (i == 31) break;

		#pragma unroll
		for (int k = 0; k < BLAKE2S_BLOCK_SIZE / 4; k++) {
			((uchar4*)(input))[k] =
				make_uchar4((A + bufidx)[4 * k], (A + bufidx)[4 * k + 1], (A + bufidx)[4 * k + 2], (A + bufidx)[4 * k + 3]);
		}
		#pragma unroll
		for (int k = 0; k < BLAKE2S_KEY_SIZE / 4; k++) {
			((uchar4*)(key))[k] =
				make_uchar4((B + bufidx)[4 * k], (B + bufidx)[4 * k + 1], (B + bufidx)[4 * k + 2], (B + bufidx)[4 * k + 3]);
		} 
		
		if (salt == NULL) Blake2S((uint32_t*)input, key);
	}

	if (salt != NULL) {
		uchar4 unfucked[1];
		unfucked[0] = make_uchar4(B[28 + bufidx], B[29 + bufidx], B[30 + bufidx], B[31 + bufidx]);
		return ((uint32_t*)unfucked)[0] ^ ((uint32_t*)A)[7];
	} else {
		#pragma nounroll
		for (int i = 0; i < FASTKDF_BUFFER_SIZE / 4; ++i) {
			((uchar4*)output)[i] =
				make_uchar4(B[(uint8_t)(4 * i + bufidx)], B[(uint8_t)(4 * i + 1 + bufidx)], B[(uint8_t)(4 * i + 2 + bufidx)], B[(uint8_t)(4 * i + 3 + bufidx)]) ^ ((uchar4*)A)[i];
		}
		return 0;
	}
}

 
#define SALSA(a,b,c,d) { \
    b^=rotate(a+d,  7); \
    c^=rotate(b+a,  9); \
    d^=rotate(c+b, 13); \
    a^=rotate(d+c, 18); \
}

#define SALSA_CORE(state) { \
	SALSA(state.s0,state.s4,state.s8,state.sc); \
	SALSA(state.s5,state.s9,state.sd,state.s1); \
	SALSA(state.sa,state.se,state.s2,state.s6); \
	SALSA(state.sf,state.s3,state.s7,state.sb); \
	SALSA(state.s0,state.s1,state.s2,state.s3); \
	SALSA(state.s5,state.s6,state.s7,state.s4); \
	SALSA(state.sa,state.sb,state.s8,state.s9); \
	SALSA(state.sf,state.sc,state.sd,state.se); \
} 

#define CHACHA_STEP(a,b,c,d) { \
	a += b; d = __byte_perm(d^a,0,0x1032); \
	c += d; b = rotate(b^c, 12); \
	a += b; d = __byte_perm(d^a,0,0x2103); \
	c += d; b = rotate(b^c, 7); \
}

#define CHACHA_CORE_PARALLEL(state)	 { \
	CHACHA_STEP(state.lo.s0, state.lo.s4, state.hi.s0, state.hi.s4); \
	CHACHA_STEP(state.lo.s1, state.lo.s5, state.hi.s1, state.hi.s5); \
	CHACHA_STEP(state.lo.s2, state.lo.s6, state.hi.s2, state.hi.s6); \
	CHACHA_STEP(state.lo.s3, state.lo.s7, state.hi.s3, state.hi.s7); \
	CHACHA_STEP(state.lo.s0, state.lo.s5, state.hi.s2, state.hi.s7); \
	CHACHA_STEP(state.lo.s1, state.lo.s6, state.hi.s3, state.hi.s4); \
	CHACHA_STEP(state.lo.s2, state.lo.s7, state.hi.s0, state.hi.s5); \
	CHACHA_STEP(state.lo.s3, state.lo.s4, state.hi.s1, state.hi.s6); \
}


__forceinline__ __device__ uint16 salsa_small_scalar_rnd(const uint16 &X)
{
	uint16 state = X;
	#pragma nounroll
	for (int i = 0; i < 10; ++i) SALSA_CORE(state);
	return(X + state);
}


__device__ __forceinline__ uint16 chacha_small_parallel_rnd(const uint16 &X)
{ 
	uint16 state = X;
	#pragma nounroll
	for (int i = 0; i < 10; ++i) CHACHA_CORE_PARALLEL(state);
	return(X + state);
}


static __device__ __forceinline__ void neoscrypt_chacha(uint16 *XV)
{
	XV[0] ^= XV[3];
	XV[0] = chacha_small_parallel_rnd(XV[0]); XV[1] ^= XV[0];
  uint16 temp = chacha_small_parallel_rnd(XV[1]); XV[2] ^= temp;
	XV[1] = chacha_small_parallel_rnd(XV[2]); XV[3] ^= XV[1];
	XV[3] = chacha_small_parallel_rnd(XV[3]);
  XV[2] = temp;
}


static __device__ __forceinline__ void neoscrypt_salsa(uint16 *XV)
{
	XV[0] ^= XV[3];
	XV[0] = salsa_small_scalar_rnd(XV[0]); XV[1] ^= XV[0];
	uint16 temp = salsa_small_scalar_rnd(XV[1]); XV[2] ^= temp;
	XV[1] = salsa_small_scalar_rnd(XV[2]); XV[3] ^= XV[1];
	XV[3] = salsa_small_scalar_rnd(XV[3]);
  XV[2] = temp;
}   

 
#define SHIFT 130

__global__
#if __CUDA_ARCH__ > 500
__launch_bounds__(128, 2)
#else
__launch_bounds__(128, 3)
#endif
void neoscrypt_gpu_hash_k0(int stratum, uint32_t startNonce)
{
	const int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const int shift = SHIFT * 16 * thread;
	const uint32_t nonce = startNonce + thread;
	
	uint16 X[4];
	uint32_t data[80];

	#pragma unroll
	for (int i = 0; i <  5; i++) ((uint4*)data)[i] = ((uint4 *)c_data)[i];	//ld.local.v4
	data[19] = (stratum) ? cuda_swab32(nonce) : nonce;
	#pragma unroll
	for (int i = 5; i < 20; i++) ((uint4*)data)[i] = ((uint4 *)data)[i % 5];

	fastkdf(data, (uint8_t*)X, NULL);	//256
	((uintx64 *)(W + shift))[0] = ((uintx64 *)X)[0];
//	((ulonglong16 *)(W + shift))[0] = ((ulonglong16 *)X)[0];
}


__global__ __launch_bounds__(128, 2) void neoscrypt_gpu_hash_k01()
{
	const int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const int shift = SHIFT * 16 * thread;
	uint16 X[4];
	((uintx64 *)X)[0]= __ldg32(&(W + shift)[0]);

	#pragma nounroll
	for (int i = 0; i < 128; ++i)
	{			
		neoscrypt_chacha(X);
//		((ulonglong16 *)(W + shift))[i + 1] = ((ulonglong16 *)X)[0];
		((uintx64 *)(W + shift))[i + 1] = ((uintx64 *)X)[0];
	}
}


__global__ __launch_bounds__(128, 2) void neoscrypt_gpu_hash_k2()
{
	const int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const int shift = SHIFT * 16 * thread;
	uint16 X[4];
	((uintx64 *)X)[0] = __ldg32(&(W + shift)[2048]);
	
	#pragma nounroll 
	for (int t = 0; t < 128; t++)
	{
		((uintx64 *)X)[0] ^= __ldg32(&(W + shift)[(X[3].lo.s0 & 0x7F) << 4]);
		neoscrypt_chacha(X);
	}
	((uintx64 *)(W + shift))[129] = ((uintx64*)X)[0];  // best checked
}


__global__ __launch_bounds__(128, 2) void neoscrypt_gpu_hash_k3()
{
	const int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const int shift = SHIFT * 16 * thread;
	uint16 Z[4];
	
	((uintx64*)Z)[0] = __ldg32(&(W + shift)[0]);

	#pragma nounroll 
	for (int i = 0; i < 128; ++i)
	{
		neoscrypt_salsa(Z);
//		((ulonglong16 *)(W + shift))[i + 1] = ((ulonglong16 *)Z)[0];
		((uintx64 *)(W + shift))[i + 1] = ((uintx64 *)Z)[0];
	}
}


__global__
#if __CUDA_ARCH__ > 500
__launch_bounds__(128, 3)
#else
__launch_bounds__(32, 12)
#endif
void neoscrypt_gpu_hash_k4(int stratum, uint32_t startNonce, uint32_t *nonceVector)
{
	const int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const uint32_t nonce = startNonce + thread;
	const int shift = SHIFT * 16 * thread;
	uint16 Z[4]; 
	uint32_t data[80];

	#pragma unroll
	for (int i = 0; i <  5; i++) ((uint4*)data)[i] = ((uint4 *)c_data)[i];
	data[19] = (stratum) ? cuda_swab32(nonce) : nonce;
	#pragma unroll
	for (int i = 5; i < 20; i++) ((uint4*)data)[i] = ((uint4 *)data)[i % 5];

	((uintx64 *)Z)[0] = __ldg32(&(W + shift)[2048]);
	#pragma nounroll
	for (int t = 0; t < 128; t++)
	{
		((uintx64 *)Z)[0] ^= __ldg32(&(W + shift)[(Z[3].lo.s0 & 0x7F) << 4]);
		neoscrypt_salsa(Z);
	}
	((uintx64 *)Z)[0] ^= __ldg32(&(W + shift)[2064]);
	
	if (fastkdf(data, NULL, (uint32_t*)Z) <= pTarget[0]) atomicCAS(&nonceVector[0], 0xffffffff, nonce);	//32
}


void neoscrypt_cpu_init(int thr_id, uint32_t *hash)
{
//	hipMemcpyToSymbol(HIP_SYMBOL(BLAKE2S_SIGMA), BLAKE2S_SIGMA_host, sizeof(BLAKE2S_SIGMA_host), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(W), &hash, sizeof(hash), 0, hipMemcpyHostToDevice);
	hipMalloc(&d_NNonce[thr_id], sizeof(uint32_t));
} 


__host__ uint32_t neoscrypt_cpu_hash_k4(int stratum, int thr_id, int threads, uint32_t startNounce, const int threadsperblock)
{
	uint32_t result;
	hipMemset(d_NNonce[thr_id], 0xffffffff, sizeof(uint32_t));

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);
 	
	neoscrypt_gpu_hash_k0  << <grid, block >> >(stratum, startNounce);  //b
	neoscrypt_gpu_hash_k01 << <grid, block >> >();  //b
	neoscrypt_gpu_hash_k2  << <grid, block >> >();  //a
	neoscrypt_gpu_hash_k3  << <grid, block >> >();  //b
	neoscrypt_gpu_hash_k4  << <grid, block >> >(stratum, startNounce, d_NNonce[thr_id]);  //a

//	MyStreamSynchronize(NULL, order, thr_id);
	hipMemcpy(&result, d_NNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
//	hipDeviceReset();
	
	return result;
}


__host__ void neoscrypt_setBlockTarget(uint32_t *pdata, const void *target)
{
	uint32_t input[16], key[16] = {0};

	((uint16*)input)[0] = ((uint16*)pdata)[0];
	((uint8*)key)[0] = ((uint8*)pdata)[0];
	Blake2Shost(input, key);

	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), ((uint32_t*) target) + 7, sizeof(uint32_t), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(input_init), input, 16 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_data), pdata, 10 * sizeof(uint64_t), 0, hipMemcpyHostToDevice);
}

