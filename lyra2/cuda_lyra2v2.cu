#include "hip/hip_runtime.h"


#include <stdio.h>
#include <memory.h>
#include "cuda_vector.h"
#define TPB52 9
#define TPB50 16

 
#define Nrow 4
#define Ncol 4
#define u64type uint2
#define vectype uint28
#define memshift 3
__device__ vectype  *DMatrix;

 
__device__ __forceinline__ void Gfunc_v35(uint2 & a, uint2 &b, uint2 &c, uint2 &d)
{

	a += b; d ^= a; d = SWAPDWORDS2(d);
	c += d; b ^= c; b = ROR24(b);
	a += b; d ^= a; d = ROR16(d);
	c += d; b ^= c; b = ROR2(b, 63);

}

__device__ __forceinline__ void round_lyra_v35(vectype* s)
{

	Gfunc_v35(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc_v35(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc_v35(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc_v35(s[0].w, s[1].w, s[2].w, s[3].w);

	Gfunc_v35(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc_v35(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc_v35(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc_v35(s[0].w, s[1].x, s[2].y, s[3].z);

}


 
__device__ __forceinline__ void reduceDuplex(vectype state[4], uint32_t thread)
{


	    vectype state1[3]; 
		uint32_t ps1 = (Nrow * Ncol * memshift * thread);
		uint32_t ps2 = (memshift * (Ncol-1) + memshift * Ncol + Nrow * Ncol * memshift * thread);

#pragma unroll 4
	for (int i = 0; i < Ncol; i++)
	{
        uint32_t s1 = ps1 + i*memshift;
        uint32_t s2 = ps2 - i*memshift;  
		
		#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix+s1)[j]); 
 
		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];
		round_lyra_v35(state); 
		#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];

		#pragma unroll
		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state1[j];

	}

}

__device__ __forceinline__ void reduceDuplex50(vectype state[4], uint32_t thread)
{
	uint32_t ps1 = (Nrow * Ncol * memshift * thread);
	uint32_t ps2 = (memshift * (Ncol - 1) + memshift * Ncol + Nrow * Ncol * memshift * thread);

#pragma unroll 4
	for (int i = 0; i < Ncol; i++)
	{
		uint32_t s1 = ps1 + i*memshift;
		uint32_t s2 = ps2 - i*memshift;

#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + s1)[j]);
		round_lyra_v35(state);

#pragma unroll
		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = __ldg4(&(DMatrix + s1)[j]) ^ state[j];

	}
}
__device__ __forceinline__ void reduceDuplexRowSetupV2(const int rowIn, const int rowInOut, const int rowOut, vectype state[4], uint32_t thread)
{


		vectype state2[3],state1[3];

		uint32_t ps1 = (memshift * Ncol * rowIn + Nrow * Ncol * memshift * thread);
		uint32_t ps2 = (memshift * Ncol * rowInOut + Nrow * Ncol * memshift * thread);
		uint32_t ps3 = (memshift * (Ncol-1) + memshift * Ncol * rowOut + Nrow * Ncol * memshift * thread);
	for (int i = 0; i < Ncol; i++)
	{
		uint32_t s1 = ps1 + i*memshift;
		uint32_t s2 = ps2 + i*memshift;
		uint32_t s3 = ps3 - i*memshift;

		#if __CUDA_ARCH__ == 500
		#pragma unroll
		for (int j = 0; j < 3; j++)
		{
			state[j] = state[j] ^ (__ldg4(&(DMatrix + s1)[j]) + __ldg4(&(DMatrix + s2)[j]));
		}
		
		round_lyra_v35(state);
#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);

#pragma unroll
		for (int j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);

#pragma unroll
		for (int j = 0; j < 3; j++) 
		{
			state1[j] ^= state[j];
			(DMatrix + s3)[j] = state1[j];
		}
		#else

#pragma unroll
		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);
#pragma unroll
		for (int j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);
#pragma unroll
		for (int j = 0; j < 3; j++)
		{
			vectype tmp = state1[j] + state2[j];
			state[j] ^= tmp;
		}


		round_lyra_v35(state);

#pragma unroll
		for (int j = 0; j < 3; j++)
		{
			state1[j] ^= state[j];
			(DMatrix + s3)[j] = state1[j];
		}

		#endif

		   ((uint2*)state2)[0] ^= ((uint2*)state)[11];
		   #pragma unroll
		   for (int j = 0; j < 11; j++)
			((uint2*)state2)[j+1] ^= ((uint2*)state)[j];


		#pragma unroll
		for (int j = 0; j < 3; j++)
		    (DMatrix + s2)[j] = state2[j];
	}


}



__device__ __forceinline__ void reduceDuplexRowtV2(const int rowIn, const int rowInOut, const int rowOut, vectype* state, uint32_t thread)
{
	int i,j;
		vectype state1[3],state2[3];
		uint32_t ps1 = (memshift * Ncol * rowIn + Nrow * Ncol * memshift * thread);
		uint32_t ps2 = (memshift * Ncol * rowInOut + Nrow * Ncol * memshift * thread);
		uint32_t ps3 = (memshift * Ncol * rowOut + Nrow * Ncol * memshift * thread);

	for (i = 0; i < Ncol; i++)
	{
		uint32_t s1 = ps1 + i*memshift;
		uint32_t s2 = ps2 + i*memshift;
		uint32_t s3 = ps3 + i*memshift;

		#pragma unroll 
		for (j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);


		#pragma unroll 
		for (j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);

		#pragma unroll 
		for (j = 0; j < 3; j++)
			          state1[j] += state2[j];

		#pragma unroll 
		for (j = 0; j < 3; j++)
			          state[j] ^= state1[j];

		round_lyra_v35(state);

		((uint2*)state2)[0] ^= ((uint2*)state)[11];
		#pragma unroll 
		for (j = 0; j < 11; j++)
		((uint2*)state2)[j + 1] ^= ((uint2*)state)[j];

#if __CUDA_ARCH__ == 500
		if (rowInOut != rowOut) 
		{
			#pragma unroll 
			for ( j = 0; j < 3; j++)
				(DMatrix + s3)[j] ^= state[j];

		} 
		if (rowInOut == rowOut)
		{
			#pragma unroll 
			for (j = 0; j < 3; j++)
			state2[j] ^= state[j];
		}
#else
		if (rowInOut != rowOut)
		{
			#pragma unroll 
			for (j = 0; j < 3; j++)
				(DMatrix + s3)[j] ^= state[j];

		} else
		{
			#pragma unroll 
			for (j = 0; j < 3; j++)
				state2[j] ^= state[j];
		}
#endif

		#pragma unroll 
		for (j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state2[j];
	}
}



#if __CUDA_ARCH__ == 500
__global__	__launch_bounds__(TPB50, 1)
#else
__global__	__launch_bounds__(TPB52, 1)
#endif
void lyra2v2_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	vectype state[4];

	uint28 blake2b_IV[2];
	if (threadIdx.x == 0) {

		((uint16*)blake2b_IV)[0] = make_uint16(
			 0xf3bcc908, 0x6a09e667 ,
			 0x84caa73b, 0xbb67ae85 ,
			 0xfe94f82b, 0x3c6ef372 ,
			 0x5f1d36f1, 0xa54ff53a ,
			 0xade682d1, 0x510e527f ,
			 0x2b3e6c1f, 0x9b05688c ,
			 0xfb41bd6b, 0x1f83d9ab ,
			 0x137e2179, 0x5be0cd19 
		);
	}

	if (thread < threads)
	{
 
		 ((uint2*)state)[0] = __ldg(&outputHash[thread]);
		 ((uint2*)state)[1] = __ldg(&outputHash[thread + threads]);
		 ((uint2*)state)[2] = __ldg(&outputHash[thread + 2 * threads]);
		 ((uint2*)state)[3] = __ldg(&outputHash[thread + 3 * threads]);

		 state[1] = state[0];

		 state[2] = ((blake2b_IV)[0]);
		 state[3] = ((blake2b_IV)[1]);

		 for (int i = 0; i<12; i++)
			 round_lyra_v35(state);
		 ((uint2*)state)[0].x ^= 0x20;
		 ((uint2*)state)[1].x ^= 0x20;
		 ((uint2*)state)[2].x ^= 0x20;
		 ((uint2*)state)[3].x ^= 0x01;
		 ((uint2*)state)[4].x ^= 0x04;
		 ((uint2*)state)[5].x ^= 0x04;
		 ((uint2*)state)[6].x ^= 0x80;
		 ((uint2*)state)[7].y ^= 0x01000000;

		 for (int i = 0; i<12; i++)
			 round_lyra_v35(state);

		uint32_t ps1 = (memshift * (Ncol - 1) + Nrow * Ncol * memshift * thread);

		for (int i = 0; i < Ncol; i++)
		{
			const uint32_t s1 = ps1 - memshift * i;
			DMatrix[s1] = state[0];
			DMatrix[s1+1] = state[1];
			DMatrix[s1+2] = state[2];
			round_lyra_v35(state);
		}

		#if __CUDA_ARCH__ == 500
			reduceDuplex50(state, thread);
		#else
			reduceDuplex50(state, thread);
		#endif



		reduceDuplexRowSetupV2(1, 0, 2, state,  thread);
		reduceDuplexRowSetupV2(2, 1, 3, state,  thread);
		uint32_t rowa;
		int prev=3;

         for (int i = 0; i < 4; i++)
        {
	     rowa = ((uint2*)state)[0].x & 3;  
		 reduceDuplexRowtV2(prev, rowa, i, state, thread);
         prev=i;
        }


		const uint32_t shift = (memshift * Ncol * rowa + Nrow * Ncol * memshift * thread);

		#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + shift)[j]);

		for (int i = 0; i < 12; i++)
        	round_lyra_v35(state);
		

		outputHash[thread]=            ((uint2*)state)[0];
		outputHash[thread + threads] = ((uint2*)state)[1];
		outputHash[thread + 2 * threads] = ((uint2*)state)[2]; 
		outputHash[thread + 3 * threads] = ((uint2*)state)[3];
//		((vectype*)outputHash)[thread] = state[0];

	} //thread
}


__host__
void lyra2v2_cpu_init(int thr_id, uint32_t threads,uint64_t *hash)
{
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix), &hash, sizeof(hash), 0, hipMemcpyHostToDevice);
}



__host__ 
void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash)
{
	uint32_t tpb;
	if (device_sm[device_map[thr_id]]==500)
		tpb = TPB50;
    else 
      tpb = TPB52;
	dim3 grid((threads + tpb - 1) / tpb);
	dim3 block(tpb);

	lyra2v2_gpu_hash_32 << <grid, block >> > (threads, startNounce, (uint2*)d_outputHash);
}

  