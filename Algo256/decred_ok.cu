#include "hip/hip_runtime.h"
/**
 * Blake-256 Decred 180-Bytes input Cuda Kernel (Tested on SM 5/5.2)
 * SP-MOD release #4
 * Tanguy Pruvot - Feb 2016
 * SP (amigaguru@gmail.com) march 2016.
 */

#include <stdint.h>
#include <memory.h>


#include <miner.h>

extern "C" {
#include <sph/sph_blake.h>
}

/* threads per block */
#define TPB 768
#define NONCES_PER_THREAD 1024

/* hash by cpu with blake 256 */
extern "C" void decred_hash(void *output, const void *input)
{
	sph_blake256_context ctx;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 180);
	sph_blake256_close(&ctx, output);
}


#include <cuda_helper.h>

#ifdef __INTELLISENSE__
#define __byte_perm(x, y, b) x
#endif

__constant__ uint32_t _ALIGN(16) d_data[27];
__constant__ uint32_t _ALIGN(16) pre[220];


/* 8 adapters max */
 uint32_t *d_resNonce[MAX_GPUS];
 uint32_t *h_resNonce[MAX_GPUS];

/* max count of found nonces in one call */
#define NBN 2
#if NBN > 1
// uint32_t extra_results[MAX_GPUS][NBN] = { UINT32_MAX };
#endif

#define ROTR32_c(x, n)	__funnelshift_r( (x), (x), (n) )	//(((x) >> (n)) | ((x) << (32 - (n))))

/* ############################################################################################################################### */

 __device__ __forceinline__ uint32_t SWAPWORDS(uint32_t value)
 {
	 ushort2 temp;
	 asm("mov.b32 {%0, %1}, %2; ": "=h"(temp.x), "=h"(temp.y) : "r"(value));
	 asm("mov.b32 %0, {%1, %2}; ": "=r"(value) : "h"(temp.y), "h"(temp.x));
	 return value;
 }

#define RSPRECHOST(x,y) { \
	prehost[i++] =(m[x] ^ u256[y]) ; \
	prehost[i++] =(m[y] ^ u256[x]); \
  }

 // __byte_perm(v[d] ^ v[a], 0, 0x1032); \
 
#define GSPREC(a,b,c,d,x,y) { \
	v[a] += v[b]+(m[x] ^ c_u256[y]) ; \
	v[d] = __byte_perm(v[d] ^ v[a], 0, 0x1032);\
	v[c] += v[d]; \
	v[b] = ROTR32_c(v[b] ^ v[c], 12); \
	v[a] +=  v[b] +(m[y] ^ c_u256[x]); \
	v[d] = __byte_perm(v[d] ^ v[a], 0, 0x0321); \
	v[c] += v[d]; \
	v[b] = ROTR32_c(v[b] ^ v[c], 7); \
}

#define GSPRECSP(a,b,c,d,x,y) { \
	v[d] = __byte_perm(v[d] ^ v[a], 0, 0x1032); \
	v[c] += v[d]; \
	v[b] = ROTR32_c(v[b] ^ v[c], 12); \
	v[a] += (m[y] ^ c_u256[x]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a], 0, 0x0321); \
	v[c] += v[d]; \
	v[b] = ROTR32_c(v[b] ^ v[c], 7); \
  }

#define GSPRECHOST(a, b, c, d, x, y) {\
	v[a] += (m[x] ^ u256[y]) + v[b]; \
	v[d] = SPH_ROTR32(v[d] ^ v[a],16); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
	v[a] += (m[y] ^ u256[x]) + v[b]; \
	v[d] = SPH_ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
	}

#define GSPREC_SP(a,b,c,d,x,y) { \
	v[a] += (pre[x]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a], 0, 0x1032); \
	v[c] += v[d]; \
	v[b] = ROTR32_c(v[b] ^ v[c], 12); \
	v[a] += (pre[y]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a], 0, 0x0321); \
	v[c] += v[d]; \
	v[b] = ROTR32_c(v[b] ^ v[c], 7); \
   }
#define GSPRECHOST(a, b, c, d, x, y) {\
	v[a] += (m[x] ^ u256[y]) + v[b]; \
	v[d] = SPH_ROTR32(v[d] ^ v[a],16); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
	v[a] += (m[y] ^ u256[x]) + v[b]; \
	v[d] = SPH_ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
 	}


/* ############################################################################################################################### */


__global__ __launch_bounds__(TPB,1)
void blake256_gpu_hash_nonce(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce)
{
	const uint32_t T0 = 180 * 8;
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
	//	const uint32_t nonce = startNonce + thread;

		const uint32_t numberofthreads = blockDim.x*gridDim.x;
		const uint32_t maxnonce = startNonce + thread + numberofthreads*NONCES_PER_THREAD - 1;
		const uint32_t threadindex = blockIdx.x*blockDim.x + threadIdx.x;
		#pragma unroll
		for (uint32_t nonce = startNonce + threadindex; nonce <= maxnonce; nonce += numberofthreads)
		{

			uint32_t v[16];

#pragma unroll 8
			for (uint32_t i = 0; i < 7; i++)
				v[i] = d_data[i];
			uint32_t backup = d_data[7];
			uint32_t m[16];

			m[0] = d_data[8];
			m[1] = d_data[9];
			m[2] = d_data[10];
			m[3] = nonce;
			v[7] = d_data[11];
#pragma unroll
			for (uint32_t i = 4; i < 13; i++)
			{
				m[i] = d_data[i + 8];
			}

			const uint32_t c_u256[16] = {
				0x243F6A88, 0x85A308D3, 0x13198A2E, 0x03707344,
				0xA4093822, 0x299F31D0, 0x082EFA98, 0xEC4E6C89,
				0x452821E6, 0x38D01377, 0xBE5466CF, 0x34E90C6C,
				0xC0AC29B7, 0xC97C50DD, 0x3F84D5B5, 0xB5470917
			};
			v[13] = c_u256[5] ^ T0;
			v[8] = d_data[21];//c_u256[0];
			v[12] = d_data[22];//c_u256[4] ^ T0;
			v[10] = d_data[23];
			v[14] = d_data[24];
			v[11] = d_data[25]; //c_u256[3];
			v[15] = d_data[26];//c_u256[7];
			v[9] = c_u256[1];


			m[21 - 8] = 0x80000001;
			m[22 - 8] = 0;
			m[23 - 8] = 0x000005a0;


			// round 1
			//		GSPREC(0, 4, 0x8, 0xC, 0, 1);
			GSPRECSP(1, 5, 0x9, 0xD, 2, 3);
			//			GSPREC_SP(1, 5, 0x9, 0xD, 2, 3);

			//		GSPREC(2, 6, 0xA, 0xE, 4, 5);
			//	GSPREC(3, 7, 0xB, 0xF, 6, 7);

			//		GSPREC(0, 5, 0xA, 0xF, 8, 9);
			v[0] += v[5];
			v[0xf] = __byte_perm(v[0xf] ^ v[0], 0, 0x1032);
			v[0xa] += v[0xf];
			v[5] = SPH_ROTR32(v[5] ^ v[0xa], 12);
			v[0] += (pre[9]) + v[5];
			v[0xf] = __byte_perm(v[0xf] ^ v[0], 0, 0x0321);
			v[0xa] += v[0xf];
			v[5] = SPH_ROTR32(v[5] ^ v[0xa], 7);

			GSPREC(1, 6, 0xB, 0xC, 10, 11);

			//		GSPREC(2, 7, 0x8, 0xD, 12, 13);
			v[0xD] = __byte_perm(v[0xD] ^ v[2], 0, 0x1032);
			v[0x8] += v[0xD];
			v[7] = SPH_ROTR32(v[7] ^ v[8], 12);
			v[2] += (pre[13]) + v[7];
			v[0xD] = __byte_perm(v[0xD] ^ v[2], 0, 0x0321);
			v[0x8] += v[0xD];
			v[7] = SPH_ROTR32(v[7] ^ v[8], 7);

			//	GSPREC(3, 4, 0x9, 0xE, 14, 15);
			v[0x9] += v[0xe];
			v[4] = SPH_ROTR32(v[4] ^ v[9], 12);
			v[3] += (pre[15]) + v[4];
			v[0xe] = __byte_perm(v[0xe] ^ v[3], 0, 0x0321);
			v[0x9] += v[0xe];
			v[4] = SPH_ROTR32(v[4] ^ v[0x9], 7);

			GSPREC_SP(0, 4, 0x8, 0xC, 16, 17);
			GSPREC_SP(1, 5, 0x9, 0xD, 18, 19);
			GSPREC_SP(2, 6, 0xA, 0xE, 20, 21);
			GSPREC_SP(3, 7, 0xB, 0xF, 22, 23);
			GSPREC_SP(0, 5, 0xA, 0xF, 24, 25);
			GSPREC_SP(1, 6, 0xB, 0xC, 26, 27);
			GSPREC_SP(2, 7, 0x8, 0xD, 28, 29);
			//			GSPREC_SP(3, 4, 0x9, 0xE, 30, 31);
			GSPREC(3, 4, 0x9, 0xE, 5, 3);


			// round 3
			GSPREC_SP(0, 4, 0x8, 0xC, 32, 33);
			GSPREC_SP(1, 5, 0x9, 0xD, 34, 35);
			GSPREC_SP(2, 6, 0xA, 0xE, 36, 37);
			GSPREC_SP(3, 7, 0xB, 0xF, 38, 39);
			GSPREC_SP(0, 5, 0xA, 0xF, 40, 41);
			//			GSPREC_SP(1, 6, 0xB, 0xC, 42, 43);
			GSPREC(1, 6, 0xB, 0xC, 3, 6);
			GSPREC_SP(2, 7, 0x8, 0xD, 44, 45);
			GSPREC_SP(3, 4, 0x9, 0xE, 46, 47);

			// round 4
			GSPREC_SP(0, 4, 0x8, 0xC, 48, 49);
			//			GSPREC_SP(1, 5, 0x9, 0xD, 50, 51);
			GSPREC(1, 5, 0x9, 0xD, 3, 1);
			GSPREC_SP(2, 6, 0xA, 0xE, 52, 53);
			GSPREC_SP(3, 7, 0xB, 0xF, 54, 55);
			GSPREC_SP(0, 5, 0xA, 0xF, 56, 57);
			GSPREC_SP(1, 6, 0xB, 0xC, 58, 59);
			GSPREC_SP(2, 7, 0x8, 0xD, 60, 61);
			GSPREC_SP(3, 4, 0x9, 0xE, 62, 63);
			// round 5
			GSPREC_SP(0, 4, 0x8, 0xC, 64, 65);
			GSPREC_SP(1, 5, 0x9, 0xD, 66, 67);
			GSPREC_SP(2, 6, 0xA, 0xE, 68, 69);
			GSPREC_SP(3, 7, 0xB, 0xF, 70, 71);
			GSPREC_SP(0, 5, 0xA, 0xF, 72, 73);
			GSPREC_SP(1, 6, 0xB, 0xC, 74, 75);
			GSPREC_SP(2, 7, 0x8, 0xD, 76, 77);
			//			GSPREC_SP(3, 4, 0x9, 0xE, 78, 79);
			GSPREC(3, 4, 0x9, 0xE, 3, 13);
			// round 6
			GSPREC_SP(0, 4, 0x8, 0xC, 80, 81);
			GSPREC_SP(1, 5, 0x9, 0xD, 82, 83);
			GSPREC_SP(2, 6, 0xA, 0xE, 84, 85);
			//		GSPREC_SP(3, 7, 0xB, 0xF, 86, 87);
			GSPREC(3, 7, 0xB, 0xF, 8, 3);
			GSPREC_SP(0, 5, 0xA, 0xF, 88, 89);
			GSPREC_SP(1, 6, 0xB, 0xC, 90, 91);
			GSPREC_SP(2, 7, 0x8, 0xD, 92, 93);
			GSPREC_SP(3, 4, 0x9, 0xE, 94, 95);
			// round 7
			GSPREC_SP(0, 4, 0x8, 0xC, 96, 97);
			GSPREC_SP(1, 5, 0x9, 0xD, 98, 99);
			GSPREC_SP(2, 6, 0xA, 0xE, 100, 101);
			GSPREC_SP(3, 7, 0xB, 0xF, 102, 103);
			GSPREC_SP(0, 5, 0xA, 0xF, 104, 105);
			//			GSPREC_SP(1, 6, 0xB, 0xC, 106, 107);
			GSPREC(1, 6, 0xB, 0xC, 6, 3);
			GSPREC_SP(2, 7, 0x8, 0xD, 108, 109);
			GSPREC_SP(3, 4, 0x9, 0xE, 110, 111);
			// round 8
			GSPREC_SP(0, 4, 0x8, 0xC, 112, 113);
			GSPREC_SP(1, 5, 0x9, 0xD, 114, 115);
			GSPREC_SP(2, 6, 0xA, 0xE, 116, 117);
			GSPREC(3, 7, 0xB, 0xF, 3, 9);
			//			GSPREC_SP(3, 7, 0xB, 0xF, 118, 119);
			GSPREC_SP(0, 5, 0xA, 0xF, 120, 121);
			GSPREC_SP(1, 6, 0xB, 0xC, 122, 123);
			GSPREC_SP(2, 7, 0x8, 0xD, 124, 125);
			GSPREC_SP(3, 4, 0x9, 0xE, 126, 127);
			// round 9
			GSPREC_SP(0, 4, 0x8, 0xC, 128, 129);
			GSPREC_SP(1, 5, 0x9, 0xD, 130, 131);
			//			GSPREC_SP(2, 6, 0xA, 0xE, 132, 133);
			GSPREC(2, 6, 0xA, 0xE, 11, 3);
			GSPREC_SP(3, 7, 0xB, 0xF, 134, 135);
			GSPREC_SP(0, 5, 0xA, 0xF, 136, 137);
			GSPREC_SP(1, 6, 0xB, 0xC, 138, 139);
			GSPREC_SP(2, 7, 0x8, 0xD, 140, 141);
			GSPREC_SP(3, 4, 0x9, 0xE, 142, 143);
			// round 10

			GSPREC_SP(0, 4, 0x8, 0xC, 144, 145);
			GSPREC_SP(1, 5, 0x9, 0xD, 146, 147);
			GSPREC_SP(2, 6, 0xA, 0xE, 148, 149);
			GSPREC_SP(3, 7, 0xB, 0xF, 150, 151);
			GSPREC_SP(0, 5, 0xA, 0xF, 152, 153);
			GSPREC_SP(1, 6, 0xB, 0xC, 154, 155);
			//			GSPREC_SP(2, 7, 0x8, 0xD, 156, 157);
			GSPREC(2, 7, 0x8, 0xD, 3, 12);
			GSPREC_SP(3, 4, 0x9, 0xE, 158, 159);

			// round 11
			/*			GSPREC(0, 4, 0x8, 0xC, 0, 1);
						GSPREC(1, 5, 0x9, 0xD, 2, 3);
						GSPREC(2, 6, 0xA, 0xE, 4, 5);
						GSPREC(3, 7, 0xB, 0xF, 6, 7);
						GSPREC(0, 5, 0xA, 0xF, 8, 9);
						GSPREC(1, 6, 0xB, 0xC, 10, 11);
						GSPREC(2, 7, 0x8, 0xD, 12, 13);
						GSPREC(3, 4, 0x9, 0xE, 14, 15);
						*/
			GSPREC_SP(0, 4, 0x8, 0xC, 160, 161);
			//			GSPREC_SP(1, 5, 0x9, 0xD, 162, 163);
			GSPREC(1, 5, 0x9, 0xD, 2, 3);
			GSPREC_SP(2, 6, 0xA, 0xE, 164, 165);
			GSPREC_SP(3, 7, 0xB, 0xF, 166, 167);
			GSPREC_SP(0, 5, 0xA, 0xF, 168, 169);
			GSPREC_SP(1, 6, 0xB, 0xC, 170, 171);
			GSPREC_SP(2, 7, 0x8, 0xD, 172, 173);
			GSPREC_SP(3, 4, 0x9, 0xE, 174, 175);


			// round 12
			GSPREC_SP(0, 4, 0x8, 0xC, 176, 177);
			GSPREC_SP(1, 5, 0x9, 0xD, 178, 179);
			GSPREC_SP(2, 6, 0xA, 0xE, 180, 181);
			GSPREC_SP(3, 7, 0xB, 0xF, 182, 183);
			GSPREC_SP(0, 5, 0xA, 0xF, 184, 185);
			GSPREC_SP(1, 6, 0xB, 0xC, 186, 187);
			GSPREC_SP(2, 7, 0x8, 0xD, 188, 189);
			//			GSPREC_SP(3, 4, 0x9, 0xE, 190, 191);
			GSPREC(3, 4, 0x9, 0xE, 5, 3);

			// round 13
			GSPREC_SP(0, 4, 0x8, 0xC, 192, 193);
			GSPREC_SP(1, 5, 0x9, 0xD, 194, 195);
			GSPREC_SP(2, 6, 0xA, 0xE, 196, 197);
			GSPREC_SP(3, 7, 0xB, 0xF, 198, 199);
			GSPREC_SP(0, 5, 0xA, 0xF, 200, 201);
			//			GSPREC_SP(1, 6, 0xB, 0xC, 202, 203);
			GSPREC(1, 6, 0xB, 0xC, 3, 6);
			GSPREC_SP(2, 7, 0x8, 0xD, 204, 205);
			GSPREC_SP(3, 4, 0x9, 0xE, 206, 207);
			// round 14
			GSPREC_SP(0, 4, 0x8, 0xC, 208, 209);
			//			GSPREC_SP(1, 5, 0x9, 0xD, 210, 211);
			GSPREC(1, 5, 0x9, 0xD, 3, 1);
			GSPREC_SP(2, 6, 0xA, 0xE, 212, 213);
			//			GSPREC(3, 7, 0xB, 0xF, 11, 14);

			v[3] += (pre[214]) + v[7];
			v[0xF] = __byte_perm(v[0xF] ^ v[3], 0, 0x1032);
			v[0xB] += v[0xF];
			v[7] = ROTR32_c(v[7] ^ v[0xB], 12);
			v[3] += (pre[215]) + v[7];
			v[0xF] = __byte_perm(v[0xF] ^ v[3], 0, 0x0321);
			v[0xB] += v[0xF];
			v[7] = ROTR32_c(v[7] ^ v[0xB], 7);

			//			GSPREC(0, 5, 0xA, 0xF, 2, 6);
			//#define GSPREC(a,b,c,d,x,y) {
			v[0] += (pre[216]) + v[5];
			v[0xF] = __byte_perm(v[0xF] ^ v[0], 0, 0x1032);
			v[0xA] += v[0xF];
			v[5] = ROTR32_c(v[5] ^ v[0xA], 12);
			v[0] += (pre[217]) + v[5];
			v[0xF] = __byte_perm(v[0xF] ^ v[0], 0, 0x0321);
			v[0xA] += v[0xF];
			//	v[5] = ROTR32_c(v[5] ^ v[0xA], 7);


			//		GSPREC(1, 6, 0xB, 0xC, 5, 10);
			//			GSPREC(2, 7, 0x8, 0xD, 4, 0);
			//#define GSPREC(2,b,c,d,x,y) { 
			v[2] += (m[4] ^ c_u256[0]) + v[7];
			v[0xD] = __byte_perm(v[0xD] ^ v[2], 0, 0x1032);
			v[0x8] += v[0xD];
			v[7] = ROTR32_c(v[7] ^ v[0x8], 12);
			v[2] += (m[0] ^ c_u256[4]) + v[7];
			v[0x8] += __byte_perm(v[0xD] ^ v[2], 0, 0x0321);
			//	v[7] = ;
			//			}
			//GSPREC(3, 4, 0x9, 0xE, 15, 8);

			//	v[7] ^= d_data[7] ^ v[15];

			if (!(ROTR32_c(v[7] ^ v[0x8], 7) ^ backup ^ v[15]))
			{
				/*	v[3] += (m[0xf] ^ c_u256[8]) + v[4];
					v[0xe] = __byte_perm(v[0xe] ^ v[3], 0, 0x1032);
					v[9] += v[0xe]; \
					v[4] = ROTR32(v[4] ^ v[9], 12);
					v[3] += (m[8] ^ c_u256[0xf]) + v[4];
					v[0xe] = __byte_perm(v[0xe] ^ v[3], 0, 0x0321);

					// only compute h6 & 7
					v[6] ^= d_data[6] ^ v[14];


					if (cuda_swab32(v[6]) <= highTarget)
					{
					*/
			/*	if (m[3] < resNonce[0])
				{
					resNonce[1] = resNonce[0];
					resNonce[0] = m[3];
				}
				else
					resNonce[1] = m[3];
				*/

				uint32_t tmp = atomicCAS(resNonce, 0xffffffff, m[3]);
				if (tmp != 0xffffffff)
					resNonce[1] = m[3];
				//}

			}
		}
	}
}

__host__ void decred_cpu_hash_nonce(const int thr_id, const uint32_t threads, const uint32_t startNonce)
{
//	uint32_t result = UINT32_MAX;

	dim3 grid((threads + TPB*NONCES_PER_THREAD - 1) / TPB / NONCES_PER_THREAD);
	dim3 block(TPB);



	/* Check error on Ctrl+C or kill to prevent segfaults on exit */
	hipMemset(d_resNonce[thr_id], 0xffffffff, NBN*sizeof(uint32_t));
	blake256_gpu_hash_nonce <<<grid, block>>> (threads, startNonce, d_resNonce[thr_id]);
	hipMemcpy(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost);
//	extra_results[thr_id][0] = h_resNonce[thr_id][0];
//	extra_results[thr_id][1] = h_resNonce[thr_id][1];
}

__host__
void decred_midstate_128(uint32_t *output, const uint32_t *input)
{
	sph_blake256_context ctx;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 128);



	memcpy(output, (void*)ctx.H, 32);
}

__host__
void decred_cpu_setBlock_52(uint32_t *penddata, const uint32_t *midstate, const uint32_t *ptarget)
{
	uint32_t _ALIGN(64) data[27];
	uint32_t _ALIGN(64) prehost[250];

//	memcpy(data, midstate, 32);
	// pre swab32
	uint32_t v[16];

	data[0] = midstate[0];
	data[1] = midstate[1];
	data[2] = midstate[2];
	data[3] = midstate[3];
	data[4] = midstate[4];
	data[5] = midstate[5];
	data[6] = midstate[6];
	data[7] = midstate[7];

	for (int i = 0; i<13; i++)
		data[8+i] = swab32(penddata[i]);
//	data[21] = 0x80000001;
//	data[22] = 0;
//	data[23] = 0x000005a0;
	const uint32_t T0 = 180 * 8;


//	for (uint32_t i = 0; i < 8; i++)
//		v[i] = data[i];

	const uint32_t u256[16] = {
		0x243F6A88, 0x85A308D3, 0x13198A2E, 0x03707344,
		0xA4093822, 0x299F31D0, 0x082EFA98, 0xEC4E6C89,
		0x452821E6, 0x38D01377, 0xBE5466CF, 0x34E90C6C,
		0xC0AC29B7, 0xC97C50DD, 0x3F84D5B5, 0xB5470917
	};

	v[0] = data[0];
	v[1] = data[1];
	v[2] = data[2];
	v[3] = data[3];
	v[4] = data[4];
	v[5] = data[5];
	v[6] = data[6];
	v[7] = data[7];




	v[8] = u256[0];
	v[9] = u256[1];
	v[10] = u256[2];
	v[11] = u256[3];

	v[12] = u256[4] ^ T0;
	v[13] = u256[5] ^ T0;
	v[14] = u256[6];
	v[15] = u256[7];

	uint32_t m[16];

	m[0] = data[8];
	m[1] = data[9];
	m[2] = data[10];
	m[3] = 0;

	for (uint32_t i = 4; i < 13; i++) {
		m[i] = data[i + 8];
	}

	m[21 - 8] = 0x80000001;
	m[22 - 8] = 0;
	m[23 - 8] = 0x000005a0;

	v[1] += (m[2] ^ u256[3]) + v[5]; 
	GSPRECHOST(0, 4, 0x8, 0xC, 0, 1);
	GSPRECHOST(2, 6, 0xA, 0xE, 4, 5);
	GSPRECHOST(3, 7, 0xB, 0xF, 6, 7);

/*	v[3] += (m[6] ^ u256[7]) + v[7];
	v[0xF] = ROTR32(v[0xF] ^ v[3], 16);
	v[0xB] += v[0xF];
	v[7] = ROTR32(v[7] ^ v[0xB], 12);
	v[3] += (m[7] ^ u256[6]) + v[7];
	v[0xF] = ROTR32(v[0xF] ^ v[3], 8);
	v[0xB] += v[0xF];
	v[7] = ROTR32(v[7] ^ v[0xB], 7);
*/

	v[0] += (m[8] ^ u256[9]);
	v[2] += (m[12] ^ u256[13]) + v[7];
	
	v[3] += (m[14] ^ u256[15]) + v[4];
	v[0xe] = ROTR32(v[0xe] ^ v[3], 16);



	data[0]=v[0];
	data[1] = v[1];
	data[2] = v[2];
	data[3] = v[3];
	data[4] = v[4];
	data[6] = v[6];
//	data[7] = v[7];
	data[11] = v[7];

	data[21] = v[8];
	data[22] = v[0xc];
	data[23] = v[0xa];
	data[24] = v[0xe];
	data[25] = v[0xb];
	data[26] = v[0xf];


	int i = 0;

	RSPRECHOST(0, 1);
	RSPRECHOST(2, 3);
	RSPRECHOST(4, 5);
	RSPRECHOST(6, 7);

	RSPRECHOST(8, 9);
	RSPRECHOST(10, 11);
	RSPRECHOST(12, 13);
	RSPRECHOST(14, 15);
	// round 2
	RSPRECHOST(14, 10);
	RSPRECHOST(4, 8);
	RSPRECHOST(9, 15);
	RSPRECHOST(13, 6);
	RSPRECHOST(1, 12);
	RSPRECHOST(0, 2);
	RSPRECHOST(11, 7);
	RSPRECHOST(5, 3);
	// round 3
	RSPRECHOST(11, 8);
	RSPRECHOST(12, 0);
	RSPRECHOST(5, 2);
	RSPRECHOST(15, 13);
	RSPRECHOST(10, 14);
	RSPRECHOST(3, 6);
	RSPRECHOST(7, 1);
	RSPRECHOST(9, 4);
	// round 4
	RSPRECHOST(7, 9);
	RSPRECHOST(3, 1);
	RSPRECHOST(13, 12);
	RSPRECHOST(11, 14);
	RSPRECHOST(2, 6);
	RSPRECHOST(5, 10);
	RSPRECHOST(4, 0);
	RSPRECHOST(15, 8);
	// round 5
	RSPRECHOST(9, 0);
	RSPRECHOST(5, 7);
	RSPRECHOST(2, 4);
	RSPRECHOST(10, 15);
	RSPRECHOST(14, 1);
	RSPRECHOST(11, 12);
	RSPRECHOST(6, 8);
	RSPRECHOST(3, 13);
	// round 6
	RSPRECHOST(2, 12);
	RSPRECHOST(6, 10);
	RSPRECHOST(0, 11);
	RSPRECHOST(8, 3);
	RSPRECHOST(4, 13);
	RSPRECHOST(7, 5);
	RSPRECHOST(15, 14);
	RSPRECHOST(1, 9);
	// round 7
	RSPRECHOST(12, 5);
	RSPRECHOST(1, 15);
	RSPRECHOST(14, 13);
	RSPRECHOST(4, 10);
	RSPRECHOST(0, 7);
	RSPRECHOST(6, 3);
	RSPRECHOST(9, 2);
	RSPRECHOST(8, 11);
	// round 8
	RSPRECHOST(13, 11);
	RSPRECHOST(7, 14);
	RSPRECHOST(12, 1);
	RSPRECHOST(3, 9);
	RSPRECHOST(5, 0);
	RSPRECHOST(15, 4);
	RSPRECHOST(8, 6);
	RSPRECHOST(2, 10);
	// round 9
	RSPRECHOST(6, 15);
	RSPRECHOST(14, 9);
	RSPRECHOST(11, 3);
	RSPRECHOST(0, 8);
	RSPRECHOST(12, 2);
	RSPRECHOST(13, 7);
	RSPRECHOST(1, 4);
	RSPRECHOST(10, 5);
	// round 10
	RSPRECHOST(10, 2);
	RSPRECHOST(8, 4);
	RSPRECHOST(7, 6);
	RSPRECHOST(1, 5);
	RSPRECHOST(15, 11);
	RSPRECHOST(9, 14);
	RSPRECHOST(3, 12);
	RSPRECHOST(13, 0);
	// round 11
	RSPRECHOST(0, 1);
	RSPRECHOST(2, 3);
	RSPRECHOST(4, 5);
	RSPRECHOST(6, 7);
	RSPRECHOST(8, 9);
	RSPRECHOST(10, 11);
	RSPRECHOST(12, 13);
	RSPRECHOST(14, 15);
	// round 12
	RSPRECHOST(14, 10);
	RSPRECHOST(4, 8);
	RSPRECHOST(9, 15);
	RSPRECHOST(13, 6);
	RSPRECHOST(1, 12);
	RSPRECHOST(0, 2);
	RSPRECHOST(11, 7);
	RSPRECHOST(5, 3);
	// round 13
	RSPRECHOST(11, 8);
	RSPRECHOST(12, 0);
	RSPRECHOST(5, 2);
	RSPRECHOST(15, 13);
	RSPRECHOST(10, 14);
	RSPRECHOST(3, 6);
	RSPRECHOST(7, 1);
	RSPRECHOST(9, 4);
	// round 14
	RSPRECHOST(7, 9);
	RSPRECHOST(3, 1);
	RSPRECHOST(13, 12);

	RSPRECHOST(11, 14);
	RSPRECHOST(2, 6);
	RSPRECHOST(5, 10);
	RSPRECHOST(4, 0);


	(hipMemcpyToSymbol(HIP_SYMBOL(d_data), data, 32 + 64 + 4 + 8, 0, hipMemcpyHostToDevice));
	(hipMemcpyToSymbol(HIP_SYMBOL(pre), prehost, 220 * 4, 0, hipMemcpyHostToDevice));
}

/* ############################################################################################################################### */

bool init[MAX_GPUS] = { 0 };

// nonce position is different in decred
#define DCR_NONCE_OFT32 35

extern "C" int scanhash_decred(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	static uint32_t _ALIGN(64) endiandata[MAX_GPUS][48];
	static uint32_t _ALIGN(64) midstate[MAX_GPUS][8];

	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	uint32_t *pnonce = &pdata[DCR_NONCE_OFT32];

	const uint32_t first_nonce = *pnonce;
//	uint64_t targetHigh = ((uint64_t*)ptarget)[3];

	int dev_id = device_map[thr_id];
	int intensity = 30;
	if (device_sm[dev_id] < 350) intensity = 22;
	if (device_sm[dev_id] > 500)
	{
		intensity = 30;
	}
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	int rc = 0;

	if (opt_benchmark) 
	{
		//targetHigh = 0x1ULL << 32;
		ptarget[6] = swab32(0xfff);
	}

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		//hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		/*	if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}
     */
		(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)));
		(hipHostMalloc(&h_resNonce[thr_id], NBN * sizeof(uint32_t)));
		init[thr_id] = true;
	}

	memcpy(endiandata[thr_id], pdata, 180);
	decred_midstate_128(midstate[thr_id], pdata); //endiandata
	decred_cpu_setBlock_52(&pdata[32], midstate[thr_id], ptarget);
	do {
		const uint32_t nonce = (*pnonce);
		// GPU HASH
		decred_cpu_hash_nonce(thr_id, throughput, nonce);
		hipDeviceSynchronize();
		if (h_resNonce[thr_id][0] != UINT32_MAX)
		{
			uint32_t vhashcpu[8];
			uint32_t Htarg = ptarget[6];

			if (opt_benchmark)
			{
				gpulog(LOG_WARNING, thr_id, "Found nonce: %0", h_resNonce[thr_id][0]);
			}

			be32enc(&endiandata[thr_id][DCR_NONCE_OFT32], h_resNonce[thr_id][0]);
			decred_hash(vhashcpu, endiandata[thr_id]);
			if (vhashcpu[6] <= Htarg && fulltest(vhashcpu, ptarget))
			{
				rc = 1;
				*hashes_done = (*pnonce) - first_nonce + throughput;
				work_set_target_ratio(work, vhashcpu);
				work->nonces[0] = swab32(h_resNonce[thr_id][0]);
#if NBN > 1
				if (h_resNonce[thr_id][1] != UINT32_MAX) {
					be32enc(&endiandata[thr_id][DCR_NONCE_OFT32], h_resNonce[thr_id][1]);
					decred_hash(vhashcpu, endiandata[thr_id]);
					if (vhashcpu[6] <= Htarg && fulltest(vhashcpu, ptarget)) {
						work->nonces[1] = swab32(h_resNonce[thr_id][1]);
						if (bn_hash_target_ratio(vhashcpu, ptarget) > work->shareratio) {
							work_set_target_ratio(work, vhashcpu);
							xchg(work->nonces[1], work->nonces[0]);
						}
						rc = 2;
					}
					h_resNonce[thr_id][1] = UINT32_MAX;
				}
#endif
				*pnonce = work->nonces[0];
				return rc;
			}
			else
			{
				if (vhashcpu[7] != 0)
				{
					applog_hash(ptarget);
					applog_compare_hash(vhashcpu, ptarget);
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", h_resNonce[1]);
				}
			}
		}

		*pnonce += throughput;

	} while (!work_restart[thr_id].restart && max_nonce > (uint64_t)throughput + (*pnonce));

	*hashes_done = (*pnonce) - first_nonce;
	return rc;
}

// cleanup
extern "C" void free_decred(int thr_id)
{
	if (!init[thr_id])
		return;

//	hipDeviceSynchronize();

	hipHostFree(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);

	init[thr_id] = false;

//	hipDeviceSynchronize();
}

